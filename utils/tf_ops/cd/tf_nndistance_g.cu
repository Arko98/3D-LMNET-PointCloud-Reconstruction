
#include <hip/hip_runtime.h>
#if GOOGLE_CUDA
#define EIGEN_USE_GPU
#include "third_party/eigen3/unsupported/Eigen/CXX11/Tensor"

__global__ void NmDistanceKernel(int b,int n,const float * xyz,int m,const float * xyz2,float * result,int * result_i){
	const int batch=512;
	__shared__ float buf[batch*2];
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		for (int k2=0;k2<m;k2+=batch){
			int end_k=min(m,k2+batch)-k2;
			for (int j=threadIdx.x;j<end_k*2;j+=blockDim.x){
				buf[j]=xyz2[(i*m+k2)*2+j];
			}
			__syncthreads();
			for (int j=threadIdx.x+blockIdx.y*blockDim.x;j<n;j+=blockDim.x*gridDim.y){
				float x1=xyz[(i*n+j)*2+0];
				float y1=xyz[(i*n+j)*2+1];
				int best_i=0;
				float best=0;
				int end_ka=end_k-(end_k&3);//
				if (end_ka==batch){
					for (int k=0;k<batch;k+=4){//
						{
							float x2=buf[k*2+0]-x1;
							float y2=buf[k*2+1]-y1;
							float d=x2*x2+y2*y2;
							if (k==0 || d<best){
								best=d;
								best_i=k+k2;
							}
						}
						{
							float x2=buf[k*2+2]-x1;
							float y2=buf[k*2+3]-y1;
							float d=x2*x2+y2*y2;
							if (d<best){
								best=d;
								best_i=k+k2+1;
							}
						}
						{
							float x2=buf[k*2+4]-x1;
							float y2=buf[k*2+5]-y1;
							float d=x2*x2+y2*y2;
							if (d<best){
								best=d;
								best_i=k+k2+2;
							}
						}
						{
							float x2=buf[k*2+6]-x1;
							float y2=buf[k*2+7]-y1;
							float d=x2*x2+y2*y2;
							if (d<best){
								best=d;
								best_i=k+k2+3;
							}
						}
					}
				}else{
					for (int k=0;k<end_ka;k+=4){//
						{
							float x2=buf[k*2+0]-x1;
							float y2=buf[k*2+1]-y1;
							float d=x2*x2+y2*y2;
							if (k==0 || d<best){
								best=d;
								best_i=k+k2;
							}
						}
						{
							float x2=buf[k*2+2]-x1;
							float y2=buf[k*2+3]-y1;
							float d=x2*x2+y2*y2;
							if (d<best){
								best=d;
								best_i=k+k2+1;
							}
						}
						{
							float x2=buf[k*2+4]-x1;
							float y2=buf[k*2+5]-y1;
							float d=x2*x2+y2*y2;
							if (d<best){
								best=d;
								best_i=k+k2+2;
							}
						}
						{
							float x2=buf[k*2+6]-x1;
							float y2=buf[k*2+7]-y1;
							float d=x2*x2+y2*y2;
							if (d<best){
								best=d;
								best_i=k+k2+3;
							}
						}
					}
				}
				for (int k=end_ka;k<end_k;k++){
					float x2=buf[k*2+0]-x1;
					float y2=buf[k*2+1]-y1;
					float d=x2*x2+y2*y2;
					if (k==0 || d<best){
						best=d;
						best_i=k+k2;
					}
				}
				if (k2==0 || result[(i*n+j)]>best){
					result[(i*n+j)]=best;
					result_i[(i*n+j)]=best_i;
				}
			}
			__syncthreads();
		}
	}
}
void NmDistanceKernelLauncher(int b,int n,const float * xyz,int m,const float * xyz2,float * result,int * result_i,float * result2,int * result2_i){
	NmDistanceKernel<<<dim3(32,16,1),512>>>(b,n,xyz,m,xyz2,result,result_i);//
	NmDistanceKernel<<<dim3(32,16,1),512>>>(b,m,xyz2,n,xyz,result2,result2_i);//
}
__global__ void NmDistanceGradKernel(int b,int n,const float * xyz1,int m,const float * xyz2,const float * grad_dist1,const int * idx1,float * grad_xyz1,float * grad_xyz2){
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		for (int j=threadIdx.x+blockIdx.y*blockDim.x;j<n;j+=blockDim.x*gridDim.y){
			float x1=xyz1[(i*n+j)*2+0];
			float y1=xyz1[(i*n+j)*2+1];
			int j2=idx1[i*n+j];
			float x2=xyz2[(i*m+j2)*2+0];
			float y2=xyz2[(i*m+j2)*2+1];
			float g=grad_dist1[i*n+j]*2;
			atomicAdd(&(grad_xyz1[(i*n+j)*2+0]),g*(x1-x2));
			atomicAdd(&(grad_xyz1[(i*n+j)*2+1]),g*(y1-y2));
			atomicAdd(&(grad_xyz2[(i*m+j2)*2+0]),-(g*(x1-x2)));
			atomicAdd(&(grad_xyz2[(i*m+j2)*2+1]),-(g*(y1-y2)));
		}
	}
}
void NmDistanceGradKernelLauncher(int b,int n,const float * xyz1,int m,const float * xyz2,const float * grad_dist1,const int * idx1,const float * grad_dist2,const int * idx2,float * grad_xyz1,float * grad_xyz2){
	cudaMemset(grad_xyz1,0,b*n*2*4);
	cudaMemset(grad_xyz2,0,b*m*2*4);
	NmDistanceGradKernel<<<dim3(1,16,1),256>>>(b,n,xyz1,m,xyz2,grad_dist1,idx1,grad_xyz1,grad_xyz2);//
	NmDistanceGradKernel<<<dim3(1,16,1),256>>>(b,m,xyz2,n,xyz1,grad_dist2,idx2,grad_xyz2,grad_xyz1);//
}

#endif